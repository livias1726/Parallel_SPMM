#include "hip/hip_runtime.h"
#include "headers/cu_hll.cuh"

__device__ Type sub_reduce(int s, Type sum){
    for(; s > 0; s >>= 1) {
        sum += __shfl_down_sync(FULL_WARP_MASK, sum, s);
    }
    return sum;
}

__device__ void spmm_ell(int rows, int start, int maxnz, const int *ja, const Type *as, const Type *x, int k, Type* y){

    extern __shared__ Type LDS[];

    const int tx = threadIdx.x, ty = threadIdx.y;
    const int bdx = blockDim.x, bdy = blockDim.y;
    const int bx = blockIdx.x, by = blockIdx.y; //const int bx = blockIdx.y, by = blockIdx.x;

    const int i = ty + (bdy * bx);  // global row id of the thread
    if (i >= rows) return;          // the last block will eventually overflow the total number of rows

    const int s = start + (ty * maxnz); // starting element of the thread
    const int tid = tx * bdy + ty;      // accumulation cell in shared memory

    /* *
     * ACCUMULATION
     *      Each thread takes the elements at (ty, tx (* bdx)), in the hack of the block,
     *      and performs the product with the element of x in column 'by'.
     *      Partial sums gets stored in a matrix like manner: LDS[tx = rows][ty = cols].
     * */
    int idx;
    LDS[tid] = 0.0;
    for (int j = tx; j < maxnz; j += bdx) { // do not break the loop when padding is reached to avoid mining warp flow
        idx = s + j;
        LDS[tid] += __dmul_rn(as[idx], x[ja[idx] * k + by]);
    }
    //__syncthreads();
    __syncwarp();

    /* *
     * REDUCTION
     *      Each row in LDS has to be reduced and, with this configuration, each warp manages 1 or more rows in LDS.
     *      To do a correct reduction, the initial offset must be set at half the x block dimension.
     * */
    LDS[tid] = sub_reduce(bdx>>1, LDS[tid]);
    //__syncthreads();
    if (tx == 0) y[i * k + by] = LDS[tid]; // let the first warp take care of the update
}

__global__ void spmm_hll_kernel(int rows, const int* maxnz, const int* hack_offset,
                                const int *ja, const Type *as, const Type *x, int k, Type* y) {

    int mnz = maxnz[blockIdx.x];
    int start = hack_offset[blockIdx.x];

    spmm_ell(rows, start, mnz, ja, as, x, k, y);
}

/*
 * Blocks are dimensioned in an inverted manner w.r.t. the logical configuration (x for the rows, y for the columns).
 * This is done to maximize warp convergence, since warps are indexed first by threadIdx.x and then threadIdx.y.
 * This configuration needs to be taken into account for the rest of the implementation.
 * */
dim3 get_block_dimensions(int m, int maxnz){
    // 2D BLOCKS
    int i, max_by, bx, by = 0;

    // find the smaller number that evenly divides WARP_SIZE that is higher than maxnz
    for (i = WARP_SIZE >> 1; i > 0; i >>= 1) {
        if (maxnz > i) {
            bx = i << 1;
            break;
        }
    }

    // each block will have at least WARP_SIZE / bx rows to have BD multiple of WARP_SIZE
    i = WARP_SIZE / bx;
    max_by = MAX_THREADS_BLOCK / bx;

    // increase by a factor of 'warpSize/blockDim.x' to increase the number of warps in the block
    while (by < m && by < max_by) { by += i; }

    return dim3(bx,by);
}

/*
 * Retrieve the maxnz value for each hack
 * */
int get_maxnz(int rows, int cols, int rb, Type* as, int *mnz){
    int i, j, new_dim, row;
    int b_ctr = 0, nz_ctr = 0, max = 0;

    int s = 0, e;
    do {
        e = MIN(rows,s+rb);

        for (i = s; i < e; i++) {
            row = i * cols;
            for (j = 0; j < cols; j++) {
                if (as[row + j] == 0) break;
                nz_ctr++;
            }
            if (nz_ctr > max) max = nz_ctr;
            nz_ctr = 0;
        }

        mnz[b_ctr] = max;       // save maxnz for block
        b_ctr++;                // increase block counter
        new_dim += max * (e-s); // compute new arrays dimension
        max = 0;
        s += rb;
    } while (s < rows);

    return new_dim;
}

/**
 * Build the HLL structure from the original ELL and the kernel dimensions.
 *
 * @param ell           original ELL structure
 * @param hll           pointer to the HLL structure to build
 * @param bdx           maximum number of rows per block
 * @param num_blocks    number of blocks to cover every row
 * */
void get_hll(ELL* ell, HLL **hll, int bdx, int num_blocks){
    int m = ell->M, maxnz = ell->MAXNZ;

    int *h_maxnz, *hack_offset, *h_ja;
    Type *h_as;
    // build HLL structure
    *hll = (HLL*) malloc(sizeof(HLL));
    h_maxnz = (int*) malloc(num_blocks * sizeof(int));
    hack_offset = (int*) malloc((num_blocks + 1) * sizeof(int));

    int dim = get_maxnz(m, maxnz, bdx, ell->AS, h_maxnz);   // populate h_maxnz and get new dimension of JA and AS
    h_ja = (int*)calloc(dim, sizeof(int));
    h_as = (Type*)calloc(dim, sizeof(Type));

    int i, j, rs, re, mnz, e_idx, h_idx = 0;
    hack_offset[0] = 0;
    // for every row block re-populate new JA and AS excluding padding overhead
    for (int nb = 0; nb < num_blocks; nb++) {
        mnz = h_maxnz[nb];
        rs = nb * bdx;
        re = MIN(m, rs+bdx);

        for (i = rs; i < re; i++) {
            for (j = 0; j < mnz; j++) {

                e_idx = (i * maxnz) + j;

                h_ja[h_idx] = ell->JA[e_idx];
                h_as[h_idx++] = ell->AS[e_idx];
            }
        }

        hack_offset[nb+1] = hack_offset[nb] + (bdx * mnz);  // populate hack offsets
    }

    // deallocate ELL
    free(ell->JA);
    free(ell->AS);
    free(ell);

    // populate HLL
    (*hll)->MAXNZ = h_maxnz;            // array of maxnz per block
    (*hll)->JA = h_ja;
    (*hll)->AS = h_as;
    (*hll)->HACK_OFFSET = hack_offset;
}

/**
 * Compute the dimensions of the kernel w.r.t. the number of rows and k and builds the HLL structure starting from
 * these dimensions and the original ELL structure.
 *
 * @param ell           original ELL structure
 * @param k             number of columns in the multi-vector
 * @param hll           pointer to the HLL structure to build
 * @param BLOCK_DIM     pointer to the block dimensions
 * @param GRID_DIM      pointer to the grid dimensions
 * @param shared_mem    pointer to the amount of shared memory
 * */
void compute_hll_dimensions(ELL* ell, int k, HLL **hll, dim3* BLOCK_DIM, dim3* GRID_DIM, int *shared_mem){

    int m = ell->M, maxnz = ell->MAXNZ;
    // 2D BLOCK :
    // (minimum between warpSize and maxnz rounded up to a divisor of warpSize) X (#rows given to the block)
    dim3 bd = get_block_dimensions(m, maxnz);
    // 2D GRID : (#blocks needed to cover A's rows) X (#columns of x)
    dim3 gd = dim3(ROUND_UP(m,bd.y), k); //dim3 gd = dim3(k, ROUND_UP(m,bd.y));

    // build the HLL structure
    get_hll(ell, hll, bd.y, gd.x); //get_hll(ell, hll, bd.y, gd.y);

    // 1D SHARED MEM treated like a matrix: 1 cell per block thread
    // cannot reach maximum shared memory thanks to limit on block size (MAX_THREADS_BLOCK * sizeof(Type) < MAX_SHM)
    *shared_mem = bd.x * bd.y * sizeof(Type);

    *BLOCK_DIM = bd;
    *GRID_DIM = gd;
}

/**
 * Allocate and transfer the structures on the device. (HLL version)
 *
 * @param hll           HLL structure previously built
 * @param num_blocks    the number of blocks that takes the different rows (x side of the grid)
 * @param d_maxnz       the array of maxnz per hack
 * @param d_hack        the array of hack offsets
 * @param d_ja          the array of column indices (ELL format)
 * @param d_as          the array of nz values (ELL format)
 * */
void alloc_cuda_hll(HLL* hll, int num_blocks, int **d_maxnz, int **d_hack, int **d_ja, Type **d_as){
    int *maxnz = hll->MAXNZ, *ja = hll->JA, *hack = hll->HACK_OFFSET;
    Type *as = hll->AS;

    int size = num_blocks * sizeof(int);
    int size_ja = hack[num_blocks] * sizeof(int);
    int size_as = hack[num_blocks] * sizeof(Type);

    /*
    int tot = size + size + size_ja + size_as;
    if (tot > MAX_GM) printf("INPUT IS TOO LARGE\n");
     */

    checkCudaErrors(hipMalloc((void**) d_maxnz, size));
    checkCudaErrors(hipMalloc((void**) d_hack, size));
    checkCudaErrors(hipMalloc((void**) d_ja, size_ja));
    checkCudaErrors(hipMalloc((void**) d_as, size_as));

    checkCudaErrors(hipMemcpy(*d_maxnz, maxnz, size, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(*d_hack, hack, size, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(*d_ja, ja, size_ja, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(*d_as, as, size_as, hipMemcpyHostToDevice));
}

void print_hll(HLL* hll){
    /*TODO*/
}