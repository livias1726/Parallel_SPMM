#include "headers/cu_utils.cuh"

void process_arguments(int argc, char** argv, FILE **f, int* k){
    if (argc < 3){
        fprintf(stderr, "Usage: %s [mm-filename] [k value]\n", argv[0]);
        exit(-1);
    }

    // create file path
    char path[PATH_MAX] = "resources/files/";
    strcat(path, argv[1]);

    //check the correct opening of the matrix file
    *f = fopen(path, "r");
    if (*f == NULL) {
        fprintf(stderr, "Cannot open '%s'\n", path);
        exit(-1);
    }

    // get k value and desired storage format
    *k = (int)strtol(argv[2], NULL, 10);
}

void alloc_cuda_csr(CSR* csr, int **d_irp, int **d_ja, Type **d_as){
    int m = csr->M;
    int nz = csr->NZ;
    int size_irp = (m+1)*sizeof(int);
    int size_ja = nz*sizeof(int);
    int size_as = nz*sizeof(Type);

    int *irp = csr->IRP, *ja = csr->JA;
    Type *as = csr->AS;

    checkCudaErrors(hipMalloc((void**) d_irp, size_irp));
    checkCudaErrors(hipMalloc((void**) d_ja, size_ja));
    checkCudaErrors(hipMalloc((void**) d_as, size_as));

    checkCudaErrors(hipMemcpy(*d_irp, irp, size_irp, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(*d_ja, ja, size_ja, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(*d_as, as, size_as, hipMemcpyHostToDevice));
}

void alloc_cuda_spmm(Type **d_x, Type **d_y, const Type *x, int m, int n, int k){

    int size_partial = k * sizeof(Type);
    int size_x = n * size_partial;

    checkCudaErrors(hipMalloc((void**) d_x, size_x));
    checkCudaErrors(hipMemcpy(*d_x, x,  size_x, hipMemcpyHostToDevice));

    checkCudaErrors(hipMalloc((void**) d_y, (m*size_partial)));
}



