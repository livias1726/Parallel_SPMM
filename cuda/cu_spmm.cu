#ifdef ELLPACK
    #include "headers/cu_hll.cuh"
#else
    #include "headers/cu_csr.cuh"
#endif

int main(int argc, char** argv) {

    // matrices
    MM_typecode t;
    FILE *f;
    int k, m, n, nz;
    Type *x, *d_x, *y_s, *y_p, *d_y;
    Type *d_as;
    int *d_ja;

    // performance
    unsigned int bytes;
    float flop, gflops_s, gflops_p, bw;
    Type abs_err, rel_err;
    StopWatchInterface *timer = 0;

    // cuda dimensioning
    dim3 BLOCK_DIM, GRID_DIM;
    int shared_mem;

#ifdef ELLPACK
    ELL *ell;   // used for serial product and as input to compute an HELL structure
    HLL *hll; // used for gpu product
    int *d_maxnz, *d_hack_offset;
#else
    CSR *csr;
    int num_blocks, *blocks;
    int *d_irp, *d_blocks;
#endif

    // -------------------------------------- Set Up ------------------------------------------- //

    // parse command line and input matrix
    process_arguments(argc, argv, &f, &k);
    process_mm(&t, f);

    // read matrix from file
    Elem** elems = read_mm(f, &m, &n, &nz, t);
    fclose(f);

    // timer
    sdkCreateTimer(&timer);

    // flops
    flop = (float)2*k*nz;

    // ------------------------------------ Memory initialization ----------------------------------- //

    alloc_struct(&x, n, k);
    alloc_struct(&y_s, m ,k);
    alloc_struct(&y_p, m ,k);

    populate_multivector(x, n, k);
    //print_matrix(x, n, k, "\nMultivector:\n");

    // convert to wanted storage format
#ifdef ELLPACK
    ell = read_mm_ell(elems, m, n, nz);
    //print_ell(ell);
#else
    csr = read_mm_csr(elems, m, n, nz);
    //print_csr(csr);
#endif

    // ------------------------------------------- Serial CPU SpMM --------------------------------------------- //

    timer->start();

#ifdef ELLPACK
    serial_product_ell(ell, x, k, y_s);
#else
    serial_product_csr(csr, x, k, y_s);
#endif

    timer->stop();

    gflops_s = (float)flop/((timer->getTime())*1.e6);
    timer->reset();

    // --------------------------------------------- GPU SpMM -------------------------------------------------- //

    // dimensioning and allocation
#ifdef ELLPACK
    compute_hll_dimensions(ell, k, &hll, &BLOCK_DIM, &GRID_DIM, &shared_mem);
    //print_hll(hll, GRID_DIM.x);
    bytes = alloc_cuda_hll(hll, GRID_DIM.x, &d_maxnz, &d_hack_offset, &d_ja, &d_as);
#else
    blocks = (int*)malloc((m+1)*sizeof(int));
    compute_csr_dimensions(csr, k, blocks, &num_blocks, &BLOCK_DIM, &GRID_DIM, &shared_mem);
    bytes = alloc_cuda_csr(csr, blocks, num_blocks, &d_irp, &d_ja, &d_as, &d_blocks);
#endif

    // to avoid bank conflicts when double values are used
    if (sizeof(Type) == 8) checkCudaErrors(hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte));
    bytes += alloc_cuda_spmm(&d_x, &d_y, x, m, n, k);

    // product
    timer->start();

#ifdef ELLPACK
    spmm_hll_kernel<<<GRID_DIM, BLOCK_DIM,shared_mem>>>(m, d_maxnz, d_hack_offset, d_ja, d_as, d_x, k, d_y);
#else
    spmm_csr_vector_kernel<<<GRID_DIM, BLOCK_DIM, shared_mem>>>(d_irp, d_ja, d_as, k, d_x, d_blocks, d_y);
#endif

    checkCudaErrors(hipDeviceSynchronize());
    timer->stop();

    gflops_p = (float)flop/((timer->getTime())*1.e6);

    // ------------------------------------------------ bandwidth computation ----------------------------------------//
    timer->start();
    checkCudaErrors(hipMemcpy(y_p, d_y, m * k * sizeof(Type), hipMemcpyDeviceToHost));
    timer->stop();

    bw = (float)bytes/((timer->getTime())*1.e6);

    // -------------------------------------------- check errors ---------------------------------------------------- //
    // --> double: relative error should be as close as possible to 2.22e−16 (IEEE double precision unit roundoff)
    // --> float: relative error should be as close as possible to 1.19e-07 (IEEE single precision unit roundoff)
    get_errors(m*k, y_s, y_p, &abs_err, &rel_err);

#ifdef SAVE
    save_result(argv[1], y_p, m, k);
#endif

    //print_matrix(y_s, m, k, "\nSerial Result:\n");
    //print_matrix(y_p, m, k, "\nParallel Result:\n");

    // ------------------------------------------- Clean up ------------------------------------------------- //
#ifdef ELLPACK
    checkCudaErrors(hipFree(d_maxnz));
    checkCudaErrors(hipFree(d_hack_offset));
    delete[] hll;
#else
    checkCudaErrors(hipFree(d_irp));
    checkCudaErrors(hipFree(d_blocks));
    delete[] csr;
#endif

    delete timer;

    checkCudaErrors(hipFree(d_ja));
    checkCudaErrors(hipFree(d_as));
    checkCudaErrors(hipFree(d_x));
    checkCudaErrors(hipFree(d_y));

    delete[] x;
    delete[] y_s;
    delete[] y_p;

    checkCudaErrors(hipDeviceReset());

    // ---------------------------------------------- Results -------------------------------------------------- //
#ifdef PERFORMANCE
    fprintf(stdout, "%f %f %.2e %.2e %f", gflops_s, gflops_p, abs_err, rel_err, bw);
#else
    fprintf(stdout, "Serial GFLOPS: %f\n"
                    "Parallel GFLOPS: %f\n"
                    "Absolute error: %.2e\n"
                    "Relative error: %.2e\n"
                    "GB/s: %f\n", gflops_s, gflops_p, abs_err, rel_err, bw);
#endif

    return 0;
}