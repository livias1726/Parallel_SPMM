#ifdef ELLPACK
    #include "headers/cu_ell.cuh"
#else
    #include "headers/cu_csr.cuh"
#endif

int main(int argc, char** argv) {

    MM_typecode t;
    FILE *f;
    int k, m, n, nz;

    double flop, gflops_s, gflops_p;
    Type abs_err, rel_err;

    Type *x, *d_x, *y_s, *y_p, *d_y;

    Type *d_as;
    int *d_ja;

    StopWatchInterface* timer = 0;
    dim3 BLOCK_DIM, GRID_DIM;
    int shared_mem;

#ifdef ELLPACK
    ELL *ell;   // used for serial product and as input to compute an HELL structure
    int maxnz;
#else
    CSR *csr;
    int num_blocks, *blocks;
    int *d_irp, *d_blocks;
#endif

    // -------------------------------------- Set Up ------------------------------------------- //

    // parse command line and input matrix
    process_arguments(argc, argv, &f, &k);
    process_mm(&t, f);

    // read matrix from file
    Elem** elems = read_mm(f, &m, &n, &nz, t);
    fclose(f);

    // timer
    sdkCreateTimer(&timer);

    // flops
    flop = (double)2*k*nz;

    // ------------------------------------ Memory initialization ----------------------------------- //

    alloc_struct(&x, n, k);
    alloc_struct(&y_s, m ,k);
    alloc_struct(&y_p, m ,k);

    populate_multivector(x, n, k);

#ifdef DEBUG
    print_matrix(x, n, k, "\nMultivector:\n");
#endif

    alloc_cuda_spmm(&d_x, &d_y, x, m, n, k);

    // convert to wanted storage format
#ifdef ELLPACK
    ell = read_mm_ell(elems, m, n, nz);
    #ifdef DEBUG
    print_ell(ell);
    #endif
#else
    csr = read_mm_csr(elems, m, n, nz);
    #ifdef DEBUG
    print_csr(csr);
    #endif
#endif

    // ------------------------------------------- Serial CPU SpMM --------------------------------------------- //

    timer->start();
#ifdef ELLPACK
    serial_product_ell(ell, x, k, y_s);
#else
    serial_product_csr(csr, x, k, y_s);
#endif
    timer->stop();

    gflops_s = (double)flop/((timer->getTime())*1.e6);
    timer->reset();

    // --------------------------------------------- GPU SpMM -------------------------------------------------- //

    // to avoid bank conflicts when double values are used
    if (!SP) checkCudaErrors(hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte));

#ifdef ELLPACK
    maxnz = ell->MAXNZ;
    compute_ell_dimensions(m, maxnz, k, &BLOCK_DIM, &GRID_DIM, &shared_mem);
    alloc_cuda_ell(ell, &d_ja, &d_as);

    // product
    timer->start();
    spmm_ell_kernel<<<GRID_DIM, BLOCK_DIM,shared_mem>>>(m, maxnz, d_ja, d_as, d_x, k, d_y);
#else
    blocks = (int*)malloc((m+1)*sizeof(int));
    compute_csr_dimensions(csr, k, blocks, &num_blocks, &BLOCK_DIM, &GRID_DIM, &shared_mem);
    alloc_cuda_csr(csr, blocks, num_blocks, &d_irp, &d_ja, &d_as, &d_blocks);

    // product
    timer->start();
    spmm_csr_vector_kernel<<<GRID_DIM, BLOCK_DIM, shared_mem>>>(d_irp, d_ja, d_as, k, d_x, d_blocks, d_y);
#endif
    checkCudaErrors(hipDeviceSynchronize());
    timer->stop();

    gflops_p = (double)flop/((timer->getTime())*1.e6);
    checkCudaErrors(hipMemcpy(y_p, d_y, m * k * sizeof(Type), hipMemcpyDeviceToHost));

    // check results
    // --> double: relative error should be as close as possible to 2.22e−16 (IEEE double precision unit roundoff)
    // --> float: relative error should be as close as possible to 1.19e-07 (IEEE single precision unit roundoff)
    get_errors(m*k, y_s, y_p, &abs_err, &rel_err);

#ifdef SAVE
    save_result(y_p, m, k);
#endif

    //print_matrix(y_s, 10, k, "\nSerial Result:\n");
    //print_matrix(y_p, 10, k, "\nParallel Result:\n");

#ifdef DEBUG
    print_matrix(y_s, m, k, "\nSerial Result:\n");
    print_matrix(y_p, m, k, "\nParallel Result:\n");
#endif

    // ------------------------------------------- Cleaning up ------------------------------------------------- //
#ifdef ELLPACK
    delete[] ell;
#else
    checkCudaErrors(hipFree(d_irp));
    checkCudaErrors(hipFree(d_blocks));
    delete[] csr;
#endif

    delete timer;

    checkCudaErrors(hipFree(d_ja));
    checkCudaErrors(hipFree(d_as));
    checkCudaErrors(hipFree(d_x));
    checkCudaErrors(hipFree(d_y));

    delete[] x;
    delete[] y_s;
    delete[] y_p;

    hipDeviceReset();

#ifdef PERFORMANCE
    fprintf(stdout, "%f %f %f %f", gflops_s, gflops_p, abs_err, rel_err);
#else
    fprintf(stdout, "Serial GFLOPS: %f\n"
                    "Parallel GFLOPS: %f\n"
                    "Absolute error: %.2e\n"
                    "Relative error: %.2e\n", gflops_s, gflops_p, abs_err, rel_err);
#endif

    return 0;
}