#ifdef ELLPACK
    #include "headers/cu_hll.cuh"
#else
    #include "headers/cu_csr.cuh"
#endif

int main(int argc, char** argv) {

    // matrices
    MM_typecode t;
    FILE *f;
    int k, m, n, nz, z;
    Type *x, *d_x, *y_s, *y_p, *d_y;
    Type *d_as;
    int *d_ja;

    // performance
    float flop, gflops_s, gflops_p;
    Type abs_err, rel_err;
    StopWatchInterface *timer = 0;

    // cuda dimensioning
    dim3 BLOCK_DIM, GRID_DIM;
    int shared_mem;

#ifdef ELLPACK
    ELL *ell;   // used for serial product and as input to compute an HELL structure
    HLL *hll; // used for gpu product
    int *d_maxnz, *d_hack_offset;
#else
    CSR *csr;
    int num_blocks, *blocks;
    int *d_irp, *d_blocks;
#endif

    // -------------------------------------- Set Up ------------------------------------------- //

    // parse command line and input matrix
    process_arguments(argc, argv, &f, &k);
    process_mm(&t, f);

    // read matrix from file
    Elem** elems = read_mm(f, &m, &n, &nz, t);
    fclose(f);

    // timer
    sdkCreateTimer(&timer);

    // flops
    flop = (float)2*k*nz;

    // ------------------------------------ Memory initialization ----------------------------------- //

    alloc_struct(&x, n, k);
    alloc_struct(&y_s, m ,k);
    alloc_struct(&y_p, m ,k);

    populate_multivector(x, n, k);
    //print_matrix(x, n, k, "\nMultivector:\n");

    // convert to wanted storage format
#ifdef ELLPACK
    ell = read_mm_ell(elems, m, n, nz);
    //print_ell(ell);
#else
    csr = read_mm_csr(elems, m, n, nz);
    //print_csr(csr);
#endif

    // ------------------------------------------- Serial CPU SpMM --------------------------------------------- //
    gflops_s = 0;
    for (z = 0; z < MAX_NUM_RUNS; ++z) {
        timer->start();
#ifdef ELLPACK
        serial_product_ell(ell, x, k, y_s);
#else
        serial_product_csr(csr, x, k, y_s);
#endif
        timer->stop();

        gflops_s += (float)flop/((timer->getTime())*1.e6);
        timer->reset();
    }

    gflops_s /= MAX_NUM_RUNS;

    // --------------------------------------------- GPU SpMM -------------------------------------------------- //

    // dimensioning and allocation
#ifdef ELLPACK
    compute_hll_dimensions(ell, k, &hll, &BLOCK_DIM, &GRID_DIM, &shared_mem);
    //print_hll(hll, GRID_DIM.x);
    alloc_cuda_hll(hll, GRID_DIM.x, &d_maxnz, &d_hack_offset, &d_ja, &d_as);
#else
    blocks = (int*)malloc((m+1)*sizeof(int));
    compute_csr_dimensions(csr, k, blocks, &num_blocks, &BLOCK_DIM, &GRID_DIM, &shared_mem);
    alloc_cuda_csr(csr, blocks, num_blocks, &d_irp, &d_ja, &d_as, &d_blocks);
#endif

    // to avoid bank conflicts when double values are used
    if (sizeof(Type) == 8) checkCudaErrors(hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte));
    alloc_cuda_spmm(&d_x, &d_y, x, m, n, k);

    // product
    gflops_p = 0;
    for (z = 0; z < MAX_NUM_RUNS; ++z) {
        timer->start();
#ifdef ELLPACK
        spmm_hll_kernel<<<GRID_DIM, BLOCK_DIM,shared_mem>>>(m, d_maxnz, d_hack_offset, d_ja, d_as, d_x, k, d_y);
#else
        spmm_csr_vector_kernel<<<GRID_DIM, BLOCK_DIM, shared_mem>>>(d_irp, d_ja, d_as, k, d_x, d_blocks, d_y);
#endif
        checkCudaErrors(hipDeviceSynchronize());
        timer->stop();

        gflops_p += (float)flop/((timer->getTime())*1.e6);
        timer->reset();
        //if (z < MAX_NUM_RUNS-1) checkCudaErrors(hipMemset(d_y, 0, m * k * sizeof(Type)));
    }

    gflops_p /= MAX_NUM_RUNS;

    checkCudaErrors(hipMemcpy(y_p, d_y, m * k * sizeof(Type), hipMemcpyDeviceToHost));

    // -------------------------------------------- check errors ---------------------------------------------------- //
    get_errors(m*k, y_s, y_p, &abs_err, &rel_err);

#ifdef SAVE
    save_result(argv[1], y_p, m, k);
#endif

    //print_matrix(y_s, m, k, "\nSerial Result:\n");
    //print_matrix(y_p, m, k, "\nParallel Result:\n");

    // ------------------------------------------- Clean up ------------------------------------------------- //
#ifdef ELLPACK
    checkCudaErrors(hipFree(d_maxnz));
    checkCudaErrors(hipFree(d_hack_offset));
    delete[] hll;
#else
    checkCudaErrors(hipFree(d_irp));
    checkCudaErrors(hipFree(d_blocks));
    delete[] csr;
#endif

    delete timer;

    checkCudaErrors(hipFree(d_ja));
    checkCudaErrors(hipFree(d_as));
    checkCudaErrors(hipFree(d_x));
    checkCudaErrors(hipFree(d_y));

    delete[] x;
    delete[] y_s;
    delete[] y_p;

    checkCudaErrors(hipDeviceReset());

    // ---------------------------------------------- Results -------------------------------------------------- //
#ifdef PERFORMANCE
    fprintf(stdout, "%f %f %.2e %.2e", gflops_s, gflops_p, abs_err, rel_err);
#else
    fprintf(stdout, "Serial GFLOPS: %f\n"
                    "Parallel GFLOPS: %f\n"
                    "Absolute error: %.2e\n"
                    "Relative error: %.2e\n", gflops_s, gflops_p, abs_err, rel_err);
#endif

    return 0;
}